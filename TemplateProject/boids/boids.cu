#include "hip/hip_runtime.h"
#include <vector>
#include <ctime>
#include <random>
#include <thread>
#include <atomic>
#include <mutex>

#include "../cudautil.cuh"
#include "../cudamem.h"

#include "boids.h"
#include "../opengl/sceneManager.h"
#include "../opengl/demos/demo_boids.h"
#include "../opengl/CoreHeaders/sceneGUI.h"


#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/quaternion.hpp>
#include <glm/gtx/quaternion.hpp>

#include <cuda_gl_interop.h>

#define BOID_COUNT (1)
#define THREADS_PER_BLOCK (256)

#define USE_SHARED_MEM
#define CHECK_VIEW_RANGE

#define VISUALIZE
//#define SIMULATE

double boidsSeparationFactor = 1.0;
double boidsCohesionFactor = 0.7;
double boidsAlignmentFactor = 1.0;
double boidsGoalFactor = 0.2;
glm::vec3 boidGoal{ 10.0f, 0.0f, 0.0f };

double boidsSeparationNeighbourhood = 1.0f;
double boidsCohesionNeighbourhood = 4.0f;
double boidsAlignmentNeighbourhood = 1.0f;

double boidsMaxVelocity = 0.01f;
double boidsViewAngle = 135.0f;

static glm::vec3 flockCenter{ 0.0f, 0.0f, 0.0f };

/// Test
double boidTestDir[3] = { 0.0, 0.0, 1.0 };

/// CUDA
static __device__ bool operator==(const float3& vec1, const float3& vec2)
{
	return vec1.x == vec2.x && vec1.y == vec2.y && vec1.z == vec2.z;
}

static __device__ float3 vecClamp(const float3& vec, float max)
{
	float len = length(vec);
	if (len != 0.0f && len > max)
	{
		return vec * (max / len);
	}
	return vec;
}
static __device__ float3 vecNormalize(const float3& vec)
{
	float len = length(vec);
	if (len == 0.0f) return vec;

	return vec / len;
}

static __device__ float3 updateSeparation(const float3& position, const float3& otherPosition, int& count, FlockConfig* config)
{
	float3 vec = position - otherPosition;
	float len = length(vec);
	if (len == 0 || len >= config->separationNeighbourhood)
	{
		return make_float3(0.0f, 0.0f, 0.0f);
	}

	count++;
	return vecNormalize(vec) / len;
}
static __device__ float3 updateCohesion(const float3& position, const float3& otherPosition, int& count, FlockConfig* config)
{
	float3 vec = position - otherPosition;
	float len = length(vec);
	if (len == 0 || len >= config->cohesionNeighbourhood)
	{
		return make_float3(0.0f, 0.0f, 0.0f);
	}

	count++;
	return otherPosition;
}
static __device__ float3 updateAlignment(const float3& position, const float3& otherPosition, const float3& otherDirection, int& count, FlockConfig* config)
{
	float3 vec = position - otherPosition;
	float len = length(vec);
	if (len == 0 || len >= config->alignmentNeighbourhood)
	{
		return make_float3(0.0f, 0.0f, 0.0f);
	}

	count++;
	return otherDirection;
}

static __device__ void updateFlock(Force& force, const float3& position, const float3& otherPosition, const float3& otherDirection, FlockConfig* config)
{
	force.separation += updateSeparation(position, otherPosition, force.separationCount, config);
	force.cohesion += updateCohesion(position, otherPosition, force.cohesionCount, config);
	force.alignment += updateAlignment(position, otherPosition, otherDirection, force.alignmentCount, config);
}
static __device__ bool isInViewRange(const float3& position, const float3& direction, const float3& otherPosition, float viewAngle)
{
#ifdef CHECK_VIEW_RANGE
	if (position == otherPosition) return false;

	float3 toTarget = vecNormalize(otherPosition - position);

	float angle = atan2(length(cross(toTarget, direction)), dot(toTarget, direction));
	return angle < viewAngle;
#else
	return true;
#endif
}

static __global__ void calculateAccelerations(Boid* __restrict__ boids, Acceleration* __restrict__ outAccelerations, const int size, FlockConfig* config)
{
#pragma region Init
#ifdef USE_SHARED_MEM
	__shared__ Boid sharedBoids[THREADS_PER_BLOCK];
#endif

	const int tileSize = blockDim.x;
	const int tileCount = gridDim.x;
	const int boidId = blockDim.x * blockIdx.x + threadIdx.x;

#ifdef USE_SHARED_MEM
	float3 position = boids[min(boidId, size - 1)].position;
	float3 direction = vecNormalize(boids[min(boidId, size - 1)].direction);
#else
	if (boidId >= size) return;

	float3 position = boids[boidId].position;
	float3 direction = vecNormalize(boids[boidId].direction);
#endif

	Force force = { 0 };

#ifdef USE_SHARED_MEM
	int boidsLeft = size;
	for (int tile = 0; tile < tileCount - 1; tile++)
	{
		int tid = tile * tileSize + threadIdx.x;
		sharedBoids[threadIdx.x] = boids[tid];
		__syncthreads();

		for (int i = 0; i < tileSize; i++)
		{
			if (isInViewRange(position, direction, sharedBoids[i].position, config->viewAngle))
			{
				updateFlock(force, position, sharedBoids[i].position, sharedBoids[i].direction, config);
			}
		}
		boidsLeft -= tileSize;
		__syncthreads();
	}
	int tid = (tileCount - 1) * tileSize + threadIdx.x;
	if (tid < size)
	{
		sharedBoids[threadIdx.x] = boids[tid];
	}
	__syncthreads();

	for (int i = 0; i < boidsLeft; i++)
	{
		if (isInViewRange(position, direction, sharedBoids[i].position, config->viewAngle))
		{
			updateFlock(force, position, sharedBoids[i].position, sharedBoids[i].direction, config);
		}
	}
	__syncthreads();

	if (boidId >= size) return;
#else
	for (int i = 0; i < size; i++)
	{
		if (isInViewRange(position, direction, boids[i].position, config->viewAngle))
		{
			updateFlock(force, position, boids[i].position, boids[i].direction, config);
		}
	}
#endif

#pragma region  Create force vector
	if (force.cohesionCount > 0)
	{
		force.cohesion /= force.cohesionCount;	// center of mass
	}
	if (force.alignmentCount > 0)
	{
		force.alignment /= force.alignmentCount;
	}
	if (force.separationCount > 0)
	{
		force.separation /= force.separationCount;
	}

	Acceleration acc;
	acc.alignment = force.alignment * config->alignmentFactor;
	acc.separation = force.separation * config->separationFactor;

	if (force.cohesionCount > 0)
	{
		acc.cohesion = (force.cohesion - position) * config->cohesionFactor;
	}
	else acc.cohesion = make_float3(0.0f, 0.0f, 0.0f);
	
	acc.goal = vecNormalize(config->goal - position) * config->goalFactor;

	outAccelerations[boidId] = acc;
#pragma endregion
}
static __global__ void calculatePositions(Boid* boids, Acceleration* accelerations, size_t size, FlockConfig* config)
{
	const int boidId = blockDim.x * blockIdx.x + threadIdx.x;
	if (boidId >= size) return;

	float3 acc = accelerations[boidId].separation + accelerations[boidId].cohesion + accelerations[boidId].alignment + accelerations[boidId].goal;

	boids[boidId].direction += acc;
	boids[boidId].direction = vecClamp(boids[boidId].direction, config->maxVelocity);
	boids[boidId].position += boids[boidId].direction;
}

/// C++
static float getInitBoidRange()
{
	return log10(BOID_COUNT);
}
static std::vector<Boid> initBoids(int count)
{
	std::random_device rd;
	std::mt19937 engine(rd());
	std::uniform_real_distribution<float> posDist(-getInitBoidRange(), getInitBoidRange());
	std::uniform_real_distribution<float> dirDist(0.01f, 0.01f);

	std::vector<Boid> boids;
	for (int i = 0; i < count; i++)
	{
		boids.emplace_back(
			make_float3(posDist(engine), posDist(engine), posDist(engine)),
			make_float3(dirDist(engine), dirDist(engine), dirDist(engine))
		);
	}

	return boids;
}
static void copyTransformsFromCuda(DemoBoids* demo, CudaMemory<Boid>& boids, CudaMemory<Acceleration>& accelerations)
{
	std::vector<Boid> cpuBoids(BOID_COUNT);
	boids.load(*cpuBoids.data(), BOID_COUNT);

	std::vector<Acceleration> cpuAccelerations(BOID_COUNT);
	accelerations.load(*cpuAccelerations.data(), BOID_COUNT);

	SceneManager* manager = SceneManager::GetInstance();
	flockCenter = glm::vec3(0.0f, 0.0f, 0.0f);

	for (int i = 0; i < cpuBoids.size(); i++)
	{
		demo->boids[i]->setTransforms(cpuBoids[i].position, cpuBoids[i].direction, cpuAccelerations[i]);
		demo->boids[i]->setViewAngle(boidsViewAngle);

		flockCenter += glm::vec3(cpuBoids[i].position.x, cpuBoids[i].position.y, cpuBoids[i].position.z);
	}

	flockCenter /= cpuBoids.size();
}

static FlockConfig update_config()
{
	FlockConfig config = { 0 };
	config.separationFactor = boidsSeparationFactor;
	config.cohesionFactor = boidsCohesionFactor;
	config.alignmentFactor = boidsAlignmentFactor;
	config.goalFactor = boidsGoalFactor;
	config.goal = make_float3(boidGoal.x, boidGoal.y, boidGoal.z);

	config.cohesionNeighbourhood = boidsCohesionNeighbourhood;
	config.separationNeighbourhood = boidsSeparationNeighbourhood;
	config.alignmentNeighbourhood = boidsAlignmentNeighbourhood;

	config.maxVelocity = boidsMaxVelocity;
	config.viewAngle = glm::radians(boidsViewAngle);

	return config;
}
static glm::vec3 getMousePos(Mouse* mouse)
{
	SceneData* sceneData = SceneManager::GetInstance()->m_sceneData;
	unsigned int* screen = SceneManager::GetInstance()->m_sceneSetting->m_screen;
	glm::vec3 position = glm::vec3(mouse->m_lastPosition[0], screen[1] - mouse->m_lastPosition[1], 1.0f);

	return glm::unProject(position, sceneData->cameras[0]->getVM(), sceneData->cameras[0]->getProjectionMatrix(), glm::vec4(0, 0, screen[0], screen[1]));
}
static void updateTarget(DemoBoids* demo)
{
	SceneData* sceneData = SceneManager::GetInstance()->m_sceneData;
	Mouse* mouse = sceneData->mouse;

	if (!mouse->clickPending) return;
	mouse->clickPending = false;

	glm::vec3 pos = getMousePos(mouse);
	glm::vec3 cameraPosition = sceneData->cameras[0]->getPosition();
	glm::vec3 toFlock = flockCenter - cameraPosition;
	glm::vec3 toTarget = glm::normalize(pos - cameraPosition);
	toTarget *= glm::dot(toFlock, toTarget);
	toTarget += cameraPosition;
		
	boidGoal = toTarget;
	demo->modelObjects[0]->setPosition(boidGoal.x, boidGoal.y, boidGoal.z);
}

static void boids_body(int argc, char** argv)
{
	srand((unsigned int) time(nullptr));

#ifdef VISUALIZE
	SceneManager* sceneManager = SceneManager::GetInstance();
	DemoBoids* demo = new DemoBoids(sceneManager->m_sceneData, BOID_COUNT);
	sceneManager->Init(argc, argv, demo);
	cudaGLSetGLDevice(0);
#endif

	std::vector<Boid> boids = initBoids(BOID_COUNT);
	CudaMemory<Boid> cudaBoids(boids.size(), boids.data());
	CudaMemory<Acceleration> cudaAccelerations(BOID_COUNT);

	dim3 blockDim(THREADS_PER_BLOCK, 1);
	dim3 gridDim(getNumberOfParts(BOID_COUNT, THREADS_PER_BLOCK), 1);

	FlockConfig flockConfig = update_config();
	CudaMemory<FlockConfig> flockConfigCuda(1, &flockConfig);

	while (true)
	{
#ifdef VISUALIZE
		flockConfigCuda.store(update_config());
#endif

#ifdef SIMULATE
		CudaTimer timer;
		timer.start();
		calculateAccelerations << <gridDim, blockDim >> > (cudaBoids.device(), cudaAccelerations.device(), BOID_COUNT, flockConfigCuda.device());
		timer.stop_wait();
#ifndef VISUALIZE
		timer.print("Update directions: ");
#endif
#endif

#ifdef SIMULATE
		timer.start();
		calculatePositions << <gridDim, blockDim >> > (cudaBoids.device(), cudaAccelerations.device(), BOID_COUNT, flockConfigCuda.device());
		timer.stop_wait();
		
#ifndef VISUALIZE
		timer.print("Update positions: ");
#endif
#endif

#ifdef VISUALIZE
		copyTransformsFromCuda(demo, cudaBoids, cudaAccelerations);
		updateTarget(demo);

		sceneManager->Refresh();

		Sleep(5);
#endif
	}
}
void boids(int argc, char** argv)
{
	boids_body(argc, argv);
}
