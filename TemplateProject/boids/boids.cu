#include "hip/hip_runtime.h"
#include <vector>
#include <ctime>
#include <random>
#include <thread>
#include <atomic>
#include <mutex>

#include "boids.h"
#include "../cudautil.cuh"
#include "../cudamem.h"
#include "../opengl/sceneManager.h"
#include "../opengl/demos/demo_boids.h"

#include <cuda_gl_interop.h>

#define BOID_COUNT (50)
#define TPB (128)

#define USE_SHARED_MEM
#define VISUALIZE


/// Structures
struct Boid
{
	Boid()
	{

	}
	Boid(float3 position, float3 direction) : position(position), direction(direction)
	{

	}

	float3 position;
	float3 direction;
};
struct Force
{
	float3 alignment;
	int alignmentCount;

	float3 cohesion;
	int cohesionCount;

	float3 separation;
	int separationCount;
};
struct FlockConfig
{
	float boidSeparationFactor = 1.0f;
	float boidCohesionFactor = 0.75f;
	float boidAlignmentFactor = 0.7f;
	float boidGoalFactor = 1 / 30.0f;

	float boidSeparateNearby = 1.0f;
	float boidCohesionNearby = 4.0f;
	float boidAlignmentNearby = 2.0f;

	float boidMaxVelocity = 0.01f;
};

static std::atomic<bool> run{ true };
static std::atomic<bool> configDirty{ true };
static std::mutex configMutex;
static FlockConfig flockConfig;

/// CUDA
__constant__ float3 cGoal;

static __device__ float vecLength(float3 vec)
{
	return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}
static __device__ float3 vecNormalize(float3 vec)
{
	float length = vecLength(vec);
	if (length == 0.0f) return vec;

	return vec / length;
}
static __device__ float3 vecClamp(float3 vec, float max)
{
	float length = vecLength(vec);
	if (length > max)
	{
		return vec * (max / length);
	}
	return vec;
}

static __device__ float3 updateSeparation(float3 position, float3 otherPosition, int& count, FlockConfig* config)
{
	float3 vec = position - otherPosition;
	float length = vecLength(vec);
	if (length == 0 || length >= config->boidSeparateNearby)
	{
		return make_float3(0.0f, 0.0f, 0.0f);
	}

	count++;
	return vecNormalize(vec) / length;
}
static __device__ float3 updateCohesion(float3 position, float3 otherPosition, int& count, FlockConfig* config)
{
	float3 vec = position - otherPosition;
	float length = vecLength(vec);
	if (length == 0 || length >= config->boidCohesionNearby)
	{
		return make_float3(0.0f, 0.0f, 0.0f);
	}

	count++;
	return otherPosition;
}
static __device__ float3 updateAlignment(float3 position, float3 otherPosition, float3 otherDirection, int& count, FlockConfig* config)
{
	float3 vec = position - otherPosition;
	float length = vecLength(vec);
	if (length == 0 || length >= config->boidAlignmentNearby)
	{
		return make_float3(0.0f, 0.0f, 0.0f);
	}

	count++;
	return otherDirection;
}

static __device__ void updateFlock(Force& force, const float3& position, const float3& otherPosition, const float3& otherDirection, FlockConfig* config)
{
	force.separation += updateSeparation(position, otherPosition, force.separationCount, config);
	force.cohesion += updateCohesion(position, otherPosition, force.cohesionCount, config);
	force.alignment += updateAlignment(position, otherPosition, otherDirection, force.alignmentCount, config);
}

static __global__ void updateDirections(Boid* __restrict__ boids, float3* __restrict__ outDirections, const int size, FlockConfig* config)
{
#pragma region Init
#ifdef USE_SHARED_MEM
	__shared__ Boid sharedBoids[TPB];
#endif

	const int tileSize = blockDim.x;
	const int tileCount = gridDim.x;
	const int boidId = blockDim.x * blockIdx.x + threadIdx.x;

#ifdef USE_SHARED_MEM
	float3 position = boids[min(boidId, size - 1)].position;
#else
	if (boidId >= size) return;

	float3 position = boids[boidId].position;
#endif
	
	Force force = { 0 };

#ifdef USE_SHARED_MEM
	int boidsLeft = size;
	for (int tile = 0; tile < tileCount - 1; tile++)
	{
		int tid = tile * tileSize + threadIdx.x;
		sharedBoids[threadIdx.x] = boids[tid];
		__syncthreads();

		for (int i = 0; i < tileSize; i++)
		{
			updateFlock(force, position, sharedBoids[i].position, sharedBoids[i].direction, config);
		}
		boidsLeft -= tileSize;
		__syncthreads();
	}
	int tid = (tileCount - 1) * tileSize + threadIdx.x;
	if (tid < size)
	{
		sharedBoids[threadIdx.x] = boids[tid];
	}
	__syncthreads();

	for (int i = 0; i < boidsLeft; i++)
	{
		updateFlock(force, position, sharedBoids[i].position, sharedBoids[i].direction, config);
	}
	__syncthreads();

	if (boidId >= size) return;
#else
	for (int i = 0; i < size; i++)
	{
		updateFlock(force, position, boids[i].position, boids[i].direction, config);
	}
#endif

#pragma region  Create force vector
	if (force.cohesionCount > 0)
	{
		force.cohesion /= force.cohesionCount;	// center of mass
	}
	if (force.alignmentCount > 0)
	{
		force.alignment /= force.alignmentCount;
	}
	if (force.separationCount > 0)
	{
		force.separation /= force.separationCount;
	}

	float3 direction = make_float3(0.0f, 0.0f, 0.0f);
	direction += force.alignment * config->boidAlignmentFactor;
	direction += force.separation * config->boidSeparationFactor;
	direction += (force.cohesion - position) * config->boidCohesionFactor;
	direction += vecNormalize(cGoal - position) * config->boidGoalFactor;

	outDirections[boidId] = direction;
#pragma endregion
}
static __global__ void updatePositions(Boid* boids, float3* directions, size_t size, FlockConfig* config)
{
	const int boidId = blockDim.x * blockIdx.x + threadIdx.x;
	if (boidId >= size) return;

	boids[boidId].direction += directions[boidId];
	boids[boidId].direction = vecClamp(boids[boidId].direction, config->boidMaxVelocity);
	boids[boidId].position += boids[boidId].direction;
}

/// C++
static std::vector<Boid> init_boids(int count)
{
	std::random_device rd;
	std::mt19937 engine(rd());
	std::uniform_real_distribution<float> posDist(0.0f, 1.0f);
	std::uniform_real_distribution<float> dirDist(0.01f, 0.01f);

	std::vector<Boid> boids;
	for (int i = 0; i < count; i++)
	{
		boids.emplace_back(
			make_float3(posDist(engine), posDist(engine), posDist(engine)),
			make_float3(dirDist(engine), dirDist(engine), dirDist(engine))
		);
	}

	return boids;
}
static void copyTransformsToCuda(DemoBoids* demo, CudaMemory<Boid>& boids)
{
	std::vector<Boid> boidsCpu(BOID_COUNT);
	boids.load(*boidsCpu.data(), BOID_COUNT);

	for (int i = 0; i < boidsCpu.size(); i++)
	{
		glm::mat4 model;
		model = glm::translate(model, glm::vec3(boidsCpu[i].position.x, boidsCpu[i].position.y, boidsCpu[i].position.z));
		model = glm::scale(model, glm::vec3(0.1f, 0.1f, 0.1f));
		demo->models[i] = model;
	}
}

void boids_body(int argc, char** argv)
{
	srand((unsigned int) time(nullptr));

#ifdef VISUALIZE
	SceneManager* sceneManager = SceneManager::GetInstance();
	DemoBoids* demo = new DemoBoids(sceneManager->m_sceneData, BOID_COUNT);
	sceneManager->Init(argc, argv, demo);
	cudaGLSetGLDevice(0);
#endif

	float3 goal = make_float3(10.0f, 0.0f, 0.0f);
	CudaConstant<float3>::toDevice(cGoal, &goal);

	std::vector<Boid> boids = init_boids(BOID_COUNT);
	CudaMemory<Boid> cudaBoids(boids.size(), boids.data());
	CudaMemory<float3> outDirectionsCuda(BOID_COUNT);

	dim3 blockDim(TPB, 1);
	dim3 gridDim(getNumberOfParts(BOID_COUNT, TPB), 1);

	CudaMemory<FlockConfig> flockConfigCuda(1, &flockConfig);

	while (run)
	{
#ifdef VISUALIZE
		{
			if (configDirty)
			{
				std::lock_guard<decltype(configMutex)> lock(configMutex);
				flockConfigCuda.store(flockConfig);
				configDirty = false;
			}
		}
#endif

		CudaTimer timer;
		timer.start();
		updateDirections << <gridDim, blockDim >> > (cudaBoids.device(), outDirectionsCuda.device(), BOID_COUNT, flockConfigCuda.device());
		timer.stop_wait();
#ifndef VISUALIZE
		timer.print("Update directions: ");
#endif

		timer.start();
		updatePositions << <gridDim, blockDim >> > (cudaBoids.device(), outDirectionsCuda.device(), BOID_COUNT, flockConfigCuda.device());
		timer.stop_wait();
		
#ifndef VISUALIZE
		timer.print("Update positions: ");
#endif

#ifdef VISUALIZE
		copyTransformsToCuda(demo, cudaBoids);
		sceneManager->Refresh();
		Sleep(5);
#endif
	}
}
void boids(int argc, char** argv)
{
	std::thread runThread(boids_body, argc, argv);
	
	std::string line;
	while (std::getline(std::cin, line))
	{
		if (line[0] == 'q')
		{
			run = false;
			runThread.join();
			break;
		}
		else if (line[0] == 'a')
		{
			std::lock_guard<decltype(configMutex)> lock(configMutex);
			configDirty = true;
			flockConfig.boidCohesionFactor = -flockConfig.boidCohesionFactor;
		}
	}
}
